#include <iostream>
#include <random>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void mult(int *a, int *b, int *c, int m1_r, int m1_c, int m2_r, int m2_c) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	c[x*m2_c + y] = 0;
	for (int i = 0; i < m1_c; i++) {
		c[x*m2_c + y] += a[m1_c*x + i] * b[m2_c*i + y];
	}
}


int main() {

	int m1_r, m1_c, m2_r, m2_c;

	cout << "Enter row col for matrix 1: ";
	cin >> m1_r >> m1_c;

	cout << "Enter row col for matrix 2: ";
	cin >> m2_r >> m2_c;

	if (m1_c != m2_r) {
		cout << "mat mul not possible" << endl;
		return 0;
	}

	int *a = new int[m1_r * m1_c];
	int *b = new int[m2_r * m2_c];
	int *c = new int[m1_r * m2_c];

	cout << "Enter matrix 1: " << endl;
	for (int i = 0; i < m1_r; i++) {
		for (int j = 0; j < m1_c; j++) {
			cin >> a[i*m1_c + j];
		}
	}

	cout << "Enter matrix 2: " << endl;
	for (int i = 0; i < m2_r; i++) {
		for (int j = 0; j < m2_c; j++) {
			cin >> b[i*m2_c + j];
		}
	}

	int *x, *y, *z;

	hipMalloc(&x, m1_r * m1_c * sizeof(int));
	hipMalloc(&y, m2_r * m2_c * sizeof(int));
	hipMalloc(&z, m1_r * m2_c * sizeof(int));

	hipMemcpy(x, a, m1_r * m1_c * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(y, b, m2_r * m2_c * sizeof(int), hipMemcpyHostToDevice);
	dim3 blocks(m1_r, m2_c);
	mult<<<blocks, 1>>>(x, y, z, m1_r, m1_c, m2_r, m2_c);
	hipMemcpy(c, z, m1_r * m2_c * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < m1_r * m1_c; i++) {
		cout << a[i] << " ";
	}
	cout << endl;
	for (int i = 0; i < m2_r * m2_c; i++) {
		cout << b[i] << " ";
	}
	cout << endl;
	for (int i = 0; i < m1_r * m2_c; i++) {
		cout << c[i] << " ";
	}
	cout << endl;
	delete[] a;
	delete[] b;
	delete[] c;

	hipFree(x);
	hipFree(y);
	hipFree(z);

	return 0;
}
